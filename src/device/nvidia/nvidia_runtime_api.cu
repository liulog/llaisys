#include "../runtime_api.hpp"
#include "hip/hip_runtime.h"

#include <cstdlib>
#include <cstring>

namespace llaisys::device::nvidia {

namespace runtime_api {
int getDeviceCount() {
    int count;
    hipError_t err = hipGetDeviceCount(&count);
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }
    return count;
}

void setDevice(int device_id) {
    hipError_t err = hipSetDevice(device_id);
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
    }
}

void deviceSynchronize() {
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
    }
}

llaisysStream_t createStream() {
    hipStream_t stream;
    hipError_t err = hipStreamCreate(&stream);
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        return nullptr;
    }
    return (llaisysStream_t) stream;
}

void destroyStream(llaisysStream_t stream) {
    hipError_t err = hipStreamDestroy((hipStream_t) stream);
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
    }
}
void streamSynchronize(llaisysStream_t stream) {
    hipError_t err = hipStreamSynchronize((hipStream_t) stream);
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
    }
}

void *mallocDevice(size_t size) {
    void *d_ptr;
    hipError_t err = hipMalloc(&d_ptr, size);
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        return nullptr;
    }
    return d_ptr;
}

void freeDevice(void *ptr) {
    hipError_t err = hipFree(ptr);
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
    }
}

void *mallocHost(size_t size) {
    void *h_ptr;
    hipError_t err = hipHostMalloc(&h_ptr, size);
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
        return nullptr;
    }
    return h_ptr;
}

void freeHost(void *ptr) {
    hipError_t err = hipHostFree(ptr);
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
    }
}

void memcpySync(void *dst, const void *src, size_t size, llaisysMemcpyKind_t kind) {
    hipError_t err;
    switch (kind) {
        case LLAISYS_MEMCPY_H2H:
            err = hipMemcpy(dst, src, size, hipMemcpyHostToHost);
            break;
        case LLAISYS_MEMCPY_H2D:
            err = hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
            break;
        case LLAISYS_MEMCPY_D2H:
            err = hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
            break;
        case LLAISYS_MEMCPY_D2D:
            err = hipMemcpy(dst, src, size, hipMemcpyDeviceToDevice);
            break;
        default:
            std::cerr << "Invalid memcpy kind!" << std::endl;
            return;
    }
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
    }
}

void memcpyAsync(void *dst, const void *src, size_t size, llaisysMemcpyKind_t kind, llaisysStream_t stream) {
    hipError_t err;
    switch (kind) {
        case LLAISYS_MEMCPY_H2H:
            err = hipMemcpyAsync(dst, src, size, hipMemcpyHostToHost, (hipStream_t)stream);
            break;
        case LLAISYS_MEMCPY_H2D:
            err = hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice, (hipStream_t)stream);
            break;
        case LLAISYS_MEMCPY_D2H:
            err = hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToHost, (hipStream_t)stream);
            break;
        case LLAISYS_MEMCPY_D2D:
            err = hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToDevice, (hipStream_t)stream);
            break;
        default:
            std::cerr << "Invalid memcpy kind!" << std::endl;
            return;
    }
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
    }
}

static const LlaisysRuntimeAPI RUNTIME_API = {
    &getDeviceCount,
    &setDevice,
    &deviceSynchronize,
    &createStream,
    &destroyStream,
    &streamSynchronize,
    &mallocDevice,
    &freeDevice,
    &mallocHost,
    &freeHost,
    &memcpySync,
    &memcpyAsync};

} // namespace runtime_api

const LlaisysRuntimeAPI *getRuntimeAPI() {
    return &runtime_api::RUNTIME_API;
}
} // namespace llaisys::device::nvidia
